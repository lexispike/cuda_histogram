/*
 * Alyxandra Spikerman
 * High Perfomance Computing
 * Homework 5 - Question 2
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// CUDA kernel
__global__ void getCount(int* values, int* histogram, int N, int CLASSES) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // we're working with a data range of 1000, so to get the class size, we need to divide the range by
    // the number of classes
    double CLASS_SIZE = 1000 / (double)CLASSES;

    for (int i = id; i < N; i += stride) {
        atomicAdd(&histogram[(int)ceil(values[i] / CLASS_SIZE) - 1], 1); // atomically add a value to the right class
    }
}

int main(int argc, char* argv[] ) {
    int N, CLASSES;
    if (argc == 3) {
        N = atoi(argv[1]);
        CLASSES = atoi(argv[2]);
        printf("N = %d\nClasses= %d\n", N, CLASSES);
    } else {
        printf("Error: must input 2 arguments, <N> <# of classes>\n");
        return 1;
    }

    // how to do cudaMalloc, cudaMemcpy supplied by provided Oak Ridge National Labs code

    size_t total_bytes = CLASSES * sizeof(int);

    // create the histogram
    int* h_histogram = (int*)malloc(total_bytes);
    int* d_histogram;
    hipMalloc(&d_histogram, total_bytes);

    for (int i = 0; i < CLASSES; i++){
        h_histogram[i] = 0; // initalize the host histogram values
    }

    size_t N_bytes = N * sizeof(int);
    srand(150);
    int* h_values = (int*)malloc(N_bytes);
    int* d_values;
    hipMalloc(&d_values, N_bytes);
    for (int i = 0; i < N; i++) {
        h_values[i] = (rand() % 1000) + 1; // calculate the values
    }

    hipMemcpy(d_histogram, h_histogram, total_bytes, hipMemcpyHostToDevice); // copy histogram to device
    hipMemcpy(d_values, h_values, N_bytes, hipMemcpyHostToDevice); // copy values to device
    time_t begin = time(NULL);
    printf("\nStart kernel\n\n");
    getCount<<< N / 128, 128 >>>(d_values, d_histogram, N, CLASSES); // Execute the kernel
    hipDeviceSynchronize(); // wait for everything to finish before accessing
    time_t end = time(NULL);

    hipMemcpy(h_histogram, d_histogram, total_bytes, hipMemcpyDeviceToHost); // Copy histogram to host

    printf("Total histogram values for %d classes\n", CLASSES);
    for (int i = 0; i < CLASSES; i++) {
        printf("Class %d: %d \n", i, h_histogram[i]);
    }

    printf("Parallel Time = %f\n", end-begin);

    // free allocated memory
    hipFree(d_values);
    hipFree(d_histogram);
    free(h_values);
    free(h_histogram);
    return 0;
}
